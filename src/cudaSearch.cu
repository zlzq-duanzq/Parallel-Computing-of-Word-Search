#include "hip/hip_runtime.h"
#include "load.h"
#include <omp.h>
using namespace std;

struct State{
    int row;
    int col;
    TrieNode *p;
    char* board;
};

struct TrieNode {
    TrieNode *children[26];
    string word;
    TrieNode() : word("") {
        for (int i = 0; i < 26; i++) {
            children[i] = nullptr;
        }
    }
};

TrieNode *buildTrie(vector<string> &words) {
    TrieNode *root = new TrieNode();

    for (int j = 0; j < words.size(); j++) {
        string word = words[j];
        TrieNode *curr = root;
        for (int i = 0; i < word.length(); i++) {
            char c = word[i] - 'a';
            if (curr->children[c] == nullptr) {
                curr->children[c] = new TrieNode();
            }
            curr = curr->children[c];
        }
        curr->word = word;
    }
    
    return root;
}

__global__ void backtrackKernel(int *boards,
        const int numBoards,
        int *emptySpaces,
        int *numEmptySpaces,
        int *finished,
        int *solved) {
        }

void cudaBacktrackKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        int *boards,
        const int numBoards,
        int *emptySpaces,
        int *numEmptySpaces,
        int *finished,
        int *solved) {

    backtrackKernel<<<blocks, threadsPerBlock>>>
        (boards, numBoards, emptySpaces, numEmptySpaces, finished, solved);
}

__global__ void cudaBFSKernel(int *old_boards,
        int *new_boards,
        int total_boards,
        int *board_index,
        int *empty_spaces,
        int *empty_space_count) {
            
        }
void callBFSKernel(const unsigned int blocks, 
                        const unsigned int threadsPerBlock,
                        int *old_boards,
                        int *new_boards,
                        int total_boards,
                        int *board_index,
                        int *empty_spaces,
                        int *empty_space_count) {
    cudaBFSKernel<<<blocks, threadsPerBlock>>>
        (old_boards, new_boards, total_boards, board_index, empty_spaces, empty_space_count);
}



int main(int argc, char** argv){
    if (argc < 8){
        printf("More arguments needed\n");
        exit(-1);
    }

    m = atoi(argv[1]);
    n = atoi(argv[2]);
    string board_filename = argv[3];
    int num_words = atoi(argv[4]);
    string word_filename = argv[5];
    
    const unsigned int threadsPerBlock = atoi(argv[6]);
    const unsigned int maxBlocks = atoi(argv[7]);

    vector<vector<char>> board(m , vector<char> (n));
    readBoard(board, board_filename);

    vector<string> words(num_words);
    readWordList(words, word_filename);

    TrieNode *root = buildTrie(words);
    
    // Initialize the CUDA timer
    float elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // TODO
    

    // Stop the CUDA timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
}